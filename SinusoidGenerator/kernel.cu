#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define FREQUENCY 1000 // Frequency in Hz
#define SAMPLE_RATE (8 * FREQUENCY)
#define N_SECONDS 4
#define N_SAMPLES (N_SECONDS * SAMPLE_RATE)

#define BLOCK_SIZE 1024

hipError_t cudaSinusoidGenerator(double *signal, unsigned int size);

__global__ void sinusoidGeneratorKernel(double *signal)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    signal[i] = sin(2*3.14159*FREQUENCY*((double)i/SAMPLE_RATE)); 
    //signal[i] = (double)i;
}

int main()
{

    double * signal = (double *)malloc(N_SAMPLES * sizeof(double));

    for (int i = 0; i < N_SAMPLES; i++)
    {
        signal[i] = (double)i;
    }

    //// Generate Sinusoid
    hipError_t cudaStatus = cudaSinusoidGenerator(signal,N_SAMPLES);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaSinusoidGenerator failed!");
        return 1;
    }

    FILE* outFile;

    outFile = fopen("signal_out.txt", "w+");
    for (int i = 0; i < N_SAMPLES; i++)
    {
        fprintf(outFile, "%f\n", signal[i]);
    }
    fclose(outFile);

    free(signal);
    

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

 // Helper function for using CUDA to generate a sinusoid
hipError_t cudaSinusoidGenerator(double *signal, unsigned int size)
{
    double *dev_signal = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_signal, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Launch a kernel on the GPU blocking out signal in grid
    dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid(N_SAMPLES/dimBlock.x, 1);
    sinusoidGeneratorKernel<<<dimGrid, dimBlock>>>(dev_signal);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(signal, dev_signal, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_signal);
    
    return cudaStatus;

}
