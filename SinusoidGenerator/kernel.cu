#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "wavfile.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_math_constants.h>



#define FREQUENCY 1000.0 // Frequency in Hz

#define SAMPLE_RATE WAVFILE_SAMPLES_PER_SECOND
#define N_SECONDS 1.0
//#define N_SAMPLES (N_SECONDS * SAMPLE_RATE)
#define N_SAMPLES N_SECONDS*(WAVFILE_SAMPLES_PER_SECOND)

#define BLOCK_SIZE 1024

hipError_t cudaSinusoidGenerator(short *signal, unsigned int size);

__global__ void sinusoidGeneratorKernel(short *signal)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    signal[i] = 10000*sin(2*HIP_PI*FREQUENCY*((short)i/SAMPLE_RATE)); 
    //signal[i] = (double)i;
}

int main()
{

    short * signal = (short *)malloc(N_SAMPLES * sizeof(short));

    for (int i = 0; i < N_SAMPLES; i++)
    {
        signal[i] = (short)i;
    }

    //// Generate Sinusoid
    hipError_t cudaStatus = cudaSinusoidGenerator(signal,N_SAMPLES);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaSinusoidGenerator failed!");
        return 1;
    }

    FILE* outFile;


    //outFile = fopen("signal_out.txt", "w+");
    /*for (int i = 0; i < N_SAMPLES; i++)
    {
        fprintf(outFile, "%f\n", signal[i]);
    }
    fclose(outFile);

    free(signal);*/

    int length = N_SAMPLES;

    outFile = wavfile_open("signal_sound.wav");
    wavfile_write(outFile, signal, length);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

 // Helper function for using CUDA to generate a sinusoid
hipError_t cudaSinusoidGenerator(short *signal, unsigned int size)
{
    short *dev_signal = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_signal, size * sizeof(short));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Launch a kernel on the GPU blocking out signal in grid
    dim3 dimBlock(BLOCK_SIZE, 1);
    dim3 dimGrid(N_SAMPLES/dimBlock.x, 1);
    sinusoidGeneratorKernel<<<dimGrid, dimBlock>>>(dev_signal);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(signal, dev_signal, size * sizeof(short), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_signal);
    
    return cudaStatus;

}
